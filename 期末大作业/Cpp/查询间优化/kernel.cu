#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <set>
#include <list>
#include <algorithm>
#include <fstream>
#include <sstream>
#include <Windows.h>
#include <hip/hip_runtime.h>
using namespace std;

vector<vector<uint32_t>> arrays;
set<uint32_t> res;
vector<bool> Flag;

bool compareBySize(uint32_t& a, uint32_t& b) {
    return arrays[a].size() != arrays[b].size() ? arrays[a].size() < arrays[b].size() : a < b;
}

bool cmp(list<uint32_t>& a, list<uint32_t>& b) {
    return a.size() < b.size();
}

__global__ void processQueries(uint32_t** d_lists, uint32_t* d_sizes, uint32_t* d_results, uint32_t num_lists, uint32_t max_len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_lists) {
        uint32_t tmp = d_lists[0][idx];
        uint32_t cnt = 1;
        for (uint32_t i = 1; i < num_lists; i++) {
            for (uint32_t j = 0; j < d_sizes[i]; j++) {
                if (d_lists[i][j] == tmp) {
                    cnt++;
                    break;
                }
            }
        }
        if (cnt == num_lists) {
            d_results[idx] = tmp;
        }
    }
}

int main() {
    ifstream file("ExpIndex", ios::binary);
    if (!file) {
        cerr << "Failed to open the file." << endl;
        return 1;
    }

    uint32_t arrayLength;
    while (file.read(reinterpret_cast<char*>(&arrayLength), sizeof(arrayLength))) {
        vector<uint32_t> array(arrayLength);
        file.read(reinterpret_cast<char*>(array.data()), arrayLength * sizeof(uint32_t));
        arrays.push_back(array);
    }
    file.close();

    ifstream queryFile("ExpQuery");
    if (!queryFile) {
        cerr << "Failed to open the query file." << endl;
        return 1;
    }

    ofstream resultFile("Baseline-2");
    if (!resultFile) {
        cerr << "Failed to create the result file." << endl;
        return 1;
    }

    LARGE_INTEGER freq, start, end0, start1, end1;
    QueryPerformanceFrequency(&freq);
    QueryPerformanceCounter(&start1);
    string line;
    int queryCount = 1;

    while (getline(queryFile, line)) {
        uint32_t k = 0;
        stringstream ss(line);
        vector<uint32_t> queryIndices;
        uint32_t row_idx;
        while (ss >> row_idx) {
            queryIndices.push_back(row_idx);
            k++;
        }
        sort(queryIndices.begin(), queryIndices.end(), compareBySize);

        vector<list<uint32_t>> lists(queryIndices.size());
        for (uint32_t i = 0; i < queryIndices.size(); i++) {
            for (uint32_t j = 0; j < arrays[queryIndices[i]].size(); j++) {
                lists[i].push_back(arrays[queryIndices[i]][j]);
            }
        }

        uint32_t max_len = 0;
        for (auto& l : lists) {
            if (l.size() > max_len) max_len = l.size();
        }

        uint32_t** h_lists = new uint32_t * [lists.size()];
        uint32_t* h_sizes = new uint32_t[lists.size()];
        uint32_t* h_results = new uint32_t[max_len];
        for (size_t i = 0; i < lists.size(); i++) {
            h_sizes[i] = lists[i].size();
            h_lists[i] = new uint32_t[lists[i].size()];
            copy(lists[i].begin(), lists[i].end(), h_lists[i]);
        }

        uint32_t** d_lists;
        uint32_t* d_sizes;
        uint32_t* d_results;
        hipMalloc((void**)&d_lists, lists.size() * sizeof(uint32_t*));
        hipMalloc((void**)&d_sizes, lists.size() * sizeof(uint32_t));
        hipMalloc((void**)&d_results, max_len * sizeof(uint32_t));

        for (size_t i = 0; i < lists.size(); i++) {
            uint32_t* d_list;
            hipMalloc((void**)&d_list, lists[i].size() * sizeof(uint32_t));
            hipMemcpy(d_list, h_lists[i], lists[i].size() * sizeof(uint32_t), hipMemcpyHostToDevice);
            hipMemcpy(d_lists + i, &d_list, sizeof(uint32_t*), hipMemcpyHostToDevice);
        }
        hipMemcpy(d_sizes, h_sizes, lists.size() * sizeof(uint32_t), hipMemcpyHostToDevice);

        QueryPerformanceCounter(&start);
        int blockSize = 256;
        int numBlocks = (max_len + blockSize - 1) / blockSize;
        processQueries << <numBlocks, blockSize >> > (d_lists, d_sizes, d_results, lists.size(), max_len);
        hipDeviceSynchronize();
        QueryPerformanceCounter(&end0);

        hipMemcpy(h_results, d_results, max_len * sizeof(uint32_t), hipMemcpyDeviceToHost);

        double elapsedSeconds = static_cast<double>(end0.QuadPart - start.QuadPart) / freq.QuadPart;
        resultFile << elapsedSeconds << endl;

        for (uint32_t i = 0; i < max_len; i++) {
            if (h_results[i] != 0) {
                res.insert(h_results[i]);
            }
        }
        queryCount++;
        res.clear();
        Flag.clear();

        for (size_t i = 0; i < lists.size(); i++) {
            hipFree(h_lists[i]);
        }
        delete[] h_lists;
        delete[] h_sizes;
        delete[] h_results;
        hipFree(d_lists);
        hipFree(d_sizes);
        hipFree(d_results);
    }

    QueryPerformanceCounter(&end1);
    double elapsedSeconds = static_cast<double>(end1.QuadPart - start1.QuadPart) / freq.QuadPart;
    resultFile << elapsedSeconds << endl;

    queryFile.close();
    resultFile.close();
    return 0;
}
